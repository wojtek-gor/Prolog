#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include <string>

int main( void ) {
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount( &count );

	for( int i = 0; i < count; i++ ) {
		hipGetDeviceProperties( &prop, i );
			printf("  Device Number                 : %d\n", i);
    		printf("  Device name                   : %s\n", prop.name);
    		printf("  Compute capability            : %d.%d\n", prop.major, prop.minor);
    		printf("  Memory Clock Rate        (KHz): %d\n", prop.memoryClockRate);
    		printf("  Memory Bus Width        (bits): %d\n", prop.memoryBusWidth);
    		printf("  Peak Memory Bandwidth   (GB/s): %.0f\n", (2.0*prop.memoryClockRate*prop.memoryBusWidth/8)/1.0e6);
    		printf("  Total global mem          (GB): %.3f\n", prop.totalGlobalMem/1024.0/1024.0/1024.0 );  	
    		printf("  Shared mem per block      (kB): %.3f\n", prop.sharedMemPerBlock/1024.0 );  	
    		printf("  Registers per block       (kB): %.3f\n", prop.regsPerBlock/1024.0 );  	
    		printf("  WARP size            (threads): %d\n", prop.warpSize );  	
    		printf("  Pitch memory              (MB): %.3f\n", prop.memPitch/1024.0/1024.0 );  	
    		printf("  Max threads per block         : %d\n", prop.maxThreadsPerBlock );  	
    		printf("  Max threads dim               : %d x %d x %d\n", prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2] );  	
    		printf("  Max grids dim                 : %d x %d x %d\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2] );  	
    		printf("  Max clock rate            (Hz): %d\n", prop.clockRate );  	
    		printf("  Total constant memory     (kB): %.3f\n", prop.totalConstMem/1024.0 );  	
    		printf("  Multi processor count         : %d\n", prop.multiProcessorCount );  	
    		printf("  Execution timeout enabled     : %d\n", prop.kernelExecTimeoutEnabled );  	
    		printf("  GPU integrated                : %d\n", prop.integrated );  	
    		printf("  Mapping host memory           : %d\n", prop.canMapHostMemory );  	
    		printf("  Compute mode                  : %d\n", prop.computeMode );  	
    		printf("  Texture 1D size               : %d\n", prop.maxTexture1D );  	
			/*
    		printf("  Texture 2D size               : %d x %d\n", prop.maxTexture2D[0],maxTexture2D[1] );  	
			printf("  Texture 3D size               : %d x %d x %d\n", prop.maxTexture3D[0],maxTexture3D[1],maxTexture3D[2] );  	
			*/
			printf("  Concurrent kernels            : %d\n", prop.concurrentKernels );  	
			printf("  ECC enabled                   : %d\n", prop.ECCEnabled );  	
			printf("  TCC driver                    : %d\n", prop.tccDriver );  	
			printf("  Unified addressing            : %d\n", prop.unifiedAddressing );  	
			printf("  L2 cache size             (kB): %.3f\n", prop.l2CacheSize/1024.0 );  	
			printf("  Max threads per multiproc     : %d\n", prop.maxThreadsPerMultiProcessor );  	
			printf("  Shared mem per multiproc  (kB): %.3f\n", prop.sharedMemPerMultiprocessor/1024.0 );  	
			printf("  Registers per multiproc   (kB): %.3f\n", prop.regsPerMultiprocessor/1024.0 );  	
			printf("  Multi GPU board               : %d\n", prop.isMultiGpuBoard );  	
			printf("  32/64 bit floats ratio        : %d\n", prop.singleToDoublePrecisionPerfRatio );  	
			printf("  Pageable memory access        : %d\n", prop.pageableMemoryAccess );  	
			printf("  Concurrent kernels            : %d\n\n", prop.concurrentManagedAccess );  
	}			
}
